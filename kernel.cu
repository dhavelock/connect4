#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include <ctime>

#include "Board.h"
#include "Player.h"

#define BLOCK_COUNT 1
#define THREAD_COUNT 1

using namespace std;

int main () {
	
    Board board (WIDTH, HEIGHT);

    //HumanPlayer player1 (RED);
	MCTSPlayer player1(RED);
    MCTSParallelPlayer player2 (BLACK);

	double duration = 0;

	while (board.getWinner() == EMPTY) {
		int move;

		if (board.getTurn() == player1.getType()) {
			clock_t start = clock();
			move = player1.makeMove(board, duration);
			printf("Seq Time : %.2fs\n", (double)(clock() - start) / CLOCKS_PER_SEC);
		}
		else {
			clock_t start = clock();
			move = player2.makeMove(board);
			duration = (double)(clock() - start) / CLOCKS_PER_SEC;
			printf("Par Time : %.2fs\n", duration);
		}

		board.makeMove(move);

		board.printBoard();
	}

	board.printBoard();

	cout << "Winner : " << board.getWinner() << endl;

    
    return 0;
}