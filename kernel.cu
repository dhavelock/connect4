#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>

#include "Board.h"
#include "Player.h"

#define BLOCK_COUNT 1
#define THREAD_COUNT 1

using namespace std;

int main () {
	
    Board board (WIDTH, HEIGHT);

    HumanPlayer player1 (RED);
    MCTSPlayer player2 (BLACK);

    while (board.getWinner() == EMPTY) {
        board.printBoard();

        int move;

        if (board.getTurn() == player1.getType()) {
            cout << endl;
            move = player1.makeMove(board);
            cout << endl;
        } else {
            cout << endl;
            move = player2.makeMove(board);
            cout << endl;
        }

        board.makeMove(move);
    }

    board.printBoard();

    cout << endl;

    cout << "Winner : " << board.getWinner() << endl;
    
    return 0;
}