#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>

#include "Board.h"
#include "Player.h"

#define BLOCK_COUNT 1
#define THREAD_COUNT 1

using namespace std;

int main () {
	
    Board board (WIDTH, HEIGHT);

    //HumanPlayer player1 (RED);
	MCTSPlayer player1(RED);
    MCTSParallelPlayer player2 (BLACK);

	while (board.getWinner() == EMPTY) {
		int move;

		if (board.getTurn() == player1.getType()) {
			move = player1.makeMove(board);
		}
		else {
			move = player2.makeMove(board);
		}

		board.makeMove(move);

		board.printBoard();
	}

	//board.printBoard();

	//cout << endl;

	cout << "Winner : " << board.getWinner() << endl;
    
    return 0;
}