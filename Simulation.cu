#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/device_functions.h>

#include "Board.h"
#include "Player.h"

#define BLOCK_COUNT 1
#define THREAD_COUNT 384
#define ROLLOUTS 50
#define NUM 4

using namespace std;

__device__ int getWinner(int boardState[HEIGHT][WIDTH]) {

	// Check Rows
	int numInRow = 0;
	int lastColor = EMPTY;
	for (int i = HEIGHT - 1; i >= 0; i--) {
		for (int j = 0; j < WIDTH; j++) {
			if (boardState[i][j] != EMPTY) {
				if (boardState[i][j] != lastColor) {
					numInRow = 0;
				}
				numInRow++;
				lastColor = boardState[i][j];
				if (numInRow == NUM) {
					return lastColor;
				}
			}
			else {
				numInRow = 0;
				lastColor = boardState[i][j];
			}
		}
		numInRow = 0;
		lastColor = EMPTY;
	}

	// Check Columns
	for (int i = 0; i < WIDTH; i++) {
		for (int j = 0; j < HEIGHT; j++) {
			if (boardState[j][i] != EMPTY && (boardState[j][i] == lastColor || (lastColor != boardState[j][i] && numInRow == 0))) {
				numInRow++;
				lastColor = boardState[j][i];
				if (numInRow == NUM) {
					return lastColor;
				}
			}
			else {
				numInRow = 0;
				lastColor = boardState[j][i];
			}
		}
		numInRow = 0;
		lastColor = EMPTY;
	}

	// Check Diagonal bottom left to top right
	for (int i = NUM - 1; i < WIDTH + HEIGHT - NUM; i++) {
		for (int j = i < HEIGHT ? 0 : i - HEIGHT + 1; j < WIDTH && i - j >= 0; j++) {
			if (boardState[i - j][j] != EMPTY) {
				if (boardState[i - j][j] != lastColor) {
					numInRow = 0;
				}
				numInRow++;
				lastColor = boardState[i - j][j];
				if (numInRow == NUM) {
					return lastColor;
				}
			}
			else {
				numInRow = 0;
				lastColor = boardState[i - j][j];
			}
		}
		numInRow = 0;
		lastColor = EMPTY;
	}

	// Check Diagonal top left to bottom right
	for (int i = NUM - WIDTH; i < HEIGHT - NUM - 1; i++) {
		for (int j = i >= 0 ? 0 : -i; j < WIDTH && i + j < HEIGHT; j++) {
			if (boardState[i + j][j] != EMPTY) {
				if (lastColor != boardState[i + j][j]) {
					numInRow = 0;
				}
				numInRow++;
				lastColor = boardState[i + j][j];
				if (numInRow == NUM) {
					return lastColor;
				}
			}
			else {
				numInRow = 0;
				lastColor = boardState[i + j][j];
			}
		}
		numInRow = 0;
		lastColor = EMPTY;
	}

	return EMPTY;
}

__device__ float generate(hiprandState* globalState, int ind)
{
	hiprandState localState = globalState[ind];
	float RANDOM = hiprand_uniform(&localState);
	globalState[ind] = localState;
	return RANDOM;
}

__global__ void setup_kernel(hiprandState* state, unsigned long seed)
{
	int id = threadIdx.x;
	hiprand_init(seed, id, 0, &state[id]);
}

__global__ void simulate(hiprandState* globalState, int boardState[HEIGHT][WIDTH], int* availableMoves, int *wins, int *totals, int numMoves, int type)
{	
	// Get Tread Id
	int id = blockIdx.x * THREAD_COUNT + threadIdx.x;
	int moveIndex = id % numMoves;

	int boardCopy[HEIGHT][WIDTH];
	for (int i = 0; i < HEIGHT; i++) {
		for (int j = 0; j < WIDTH; j++) {
			boardCopy[i][j] = boardState[i][j];
		}
	}

	int plays = 0;
	int win = 0;

	// Simulate Games
	while(plays < ROLLOUTS) {

		// Create Board Copy
		for (int i = 0; i < HEIGHT; i++) {
			for (int j = 0; j < WIDTH; j++) {
				boardCopy[i][j] = boardState[i][j];
			}
		}

		int number = generate(globalState, id) * 1000000;
		int move = number % numMoves;

		// Make Move
		int success = false;
		for (int i = HEIGHT - 1; i >= 0; i--) {
			if (boardCopy[i][availableMoves[moveIndex]] == EMPTY) {
				boardCopy[i][availableMoves[moveIndex]] = type;
				success = true;
				break;
			}
		}

		if (!success) printf("FAILED TO MAKE MOVE : moveIndex %d : %d\n", moveIndex, availableMoves[moveIndex]);

		// Perform Random Playout
		int winner = getWinner(boardCopy);

		number = generate(globalState, id) * 1000000;
		int moveAvail = true;
		int turn = 1 - type;
		int count = 0;
		while (winner == EMPTY) {

			// Check if game is over
			bool gameover = true;
			for (int i = 0; i < WIDTH; i++) {
				if (boardCopy[0][i] == EMPTY) {
					gameover = false;
					break;
				}
			}
			if (gameover) {
				break;
			}

			// Make random move
			number = generate(globalState, id) * 1000000;
			int randomMove = availableMoves[number % numMoves];
			int randomSuccess = false;
			while (!randomSuccess) {
				randomSuccess = false;
				for (int i = HEIGHT - 1; i >= 0; i--) {
					if (boardCopy[i][randomMove] == EMPTY) {
						boardCopy[i][randomMove] = turn;
						turn = 1 - turn;
						randomSuccess = true;
						break;
					}
				}
				number = generate(globalState, id) * 1000000;
				randomMove = availableMoves[number % numMoves];
			}
			
			winner = getWinner(boardCopy);
		}

		if (getWinner(boardCopy) == type) win++;

		plays++;
	}

	atomicAdd(&(wins[moveIndex]), win);
	atomicAdd(&(totals[moveIndex]), plays);

}

int runSimulation(Board board, int t)
{
	// Create copy of current board state
	vector<vector<int> > boardState = board.getBoardState();
	int boardArr[HEIGHT][WIDTH];
	for (int i = 0; i < HEIGHT; i++) {
		for (int j = 0; j < WIDTH; j++) {
			boardArr[i][j] = boardState[i][j];
		}
	}
	int * boardStateDevice;
	int* availMovesDevice, *wins, *totals;
	int numAvailMoves = board.getLegalMoves().size();

	// Allocate device memory
	hipMallocManaged((void**)& boardStateDevice, HEIGHT * WIDTH * sizeof(int));
	hipMallocManaged((void**)& availMovesDevice, numAvailMoves * sizeof(int));
	hipMallocManaged((void**)& wins, numAvailMoves * sizeof(int));
	hipMallocManaged((void**)& totals, numAvailMoves * sizeof(int));

	// Copy values to board and availMoves
	std::copy(&boardArr[0][0], &boardArr[0][0] + HEIGHT * WIDTH, boardStateDevice);
	
	for (int i = 0; i < numAvailMoves; i++) {
		availMovesDevice[i] = board.getLegalMoves()[i];
		wins[i] = 0;
		totals[i] = 0;
	}

	// Random Number Generator
	srand(time(0));
	hiprandState* devStates;
	hipMalloc(&devStates, numAvailMoves * sizeof(hiprandState));
	int seed = rand();

	// Start kernel threads
	setup_kernel << <BLOCK_COUNT, THREAD_COUNT >> > (devStates, seed);

	simulate << <BLOCK_COUNT, THREAD_COUNT >> > (devStates, reinterpret_cast<int(*)[WIDTH]>(boardStateDevice), availMovesDevice, wins, totals, numAvailMoves, t);
	
	hipDeviceSynchronize();

	int move = 0;
	int maxWins = 0;
	int maxTotals = 1;
	for (int i = 0; i < numAvailMoves; i++) {
		printf("%d : %f : %d / %d\n", availMovesDevice[i], (float)wins[i] / (float)totals[i], wins[i], totals[i]);
		if (((float)wins[i] / (float)totals[i]) >= ((float)maxWins / (float)maxTotals)) {
			maxWins = wins[i];
			maxTotals = totals[i];
			move = availMovesDevice[i];
		}
	}

	// Free memory
	hipFree(wins);
	hipFree(totals);
	hipFree(devStates);
	hipFree(boardStateDevice);
	hipFree(availMovesDevice);

	printf("Parallel : %d\n", move);

	return move;
}